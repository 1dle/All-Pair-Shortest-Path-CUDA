#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"

#define BLOCK_SIZE 32

int n, m, block_size;
int *r_dist;

__global__
void kernel_phase1(int round, int n, int* dist)
{
    __shared__ int shared_dist[BLOCK_SIZE][BLOCK_SIZE];

    int x = threadIdx.x,
        y = threadIdx.y,
        i = x + round * BLOCK_SIZE,
        j = y + round * BLOCK_SIZE;

    shared_dist[x][y] = (i < n && j < n) ? dist[i * n + j] : INF;
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
        int tmp = shared_dist[x][k] + shared_dist[k][y];
        if (tmp < shared_dist[x][y]) shared_dist[x][y] = tmp;
        __syncthreads();
    }
    if (i < n && j < n) dist[i * n + j] = shared_dist[x][y];
}

__global__
void kernel_phase2(int round, int n, int* dist)
{
    if (blockIdx.x == round) return;

    __shared__ int shared_pivot[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_dist[BLOCK_SIZE][BLOCK_SIZE];

    int x = threadIdx.x,
        y = threadIdx.y,
        i = x + round * BLOCK_SIZE,
        j = y + round * BLOCK_SIZE;

    shared_pivot[x][y] = (i < n && j < n) ? dist[i * n + j] : INF;

    if (blockIdx.y == 0)
        j = y + blockIdx.x * BLOCK_SIZE;
    else
        i = x + blockIdx.x * BLOCK_SIZE;

    shared_dist[x][y] = (i < n && j < n) ? dist[i * n + j] : INF;
    __syncthreads();

    if (blockIdx.y == 1) {
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            int tmp = shared_dist[x][k] + shared_pivot[k][y];
            if (tmp < shared_dist[x][y]) shared_dist[x][y] = tmp;
        }
    } else {
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            int tmp = shared_pivot[x][k] + shared_dist[k][y];
            if (tmp < shared_dist[x][y]) shared_dist[x][y] = tmp;
        }
    }

    if (i < n && j < n) dist[i * n + j] = shared_dist[x][y];
}

__global__
void kernel_phase3(int round, int n, int* dist)
{
    if (blockIdx.x == round || blockIdx.y == round) return;

    __shared__ int shared_pivot_row[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_pivot_col[BLOCK_SIZE][BLOCK_SIZE];

    int x = threadIdx.x,
        y = threadIdx.y,
        i = x + blockIdx.x * blockDim.x,
        j = y + blockIdx.y * blockDim.y,
        i_col = y + round * BLOCK_SIZE,
        j_row = x + round * BLOCK_SIZE;

    shared_pivot_row[x][y] = (i < n && i_col < n) ? dist[i * n + i_col] : INF;
    shared_pivot_col[x][y] = (j < n && j_row < n) ? dist[j_row * n + j] : INF;
    __syncthreads();

    if (i >= n || j >= n) return;
    int dij = dist[i * n + j];
    #pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
        int tmp = shared_pivot_row[x][k] + shared_pivot_col[k][y];
        if (tmp < dij) dij = tmp;
    }
    dist[i * n + j] = dij;
}

void block_FW(int block_size)
{
    float k_time;

    int *device_dist;
    int round = (n + block_size - 1) / block_size;
    ssize_t sz = sizeof(int) * n * n;
    hipEvent_t start, stop;

    hipSetDevice(1);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipMalloc(&device_dist, sz);
    hipMemcpy(device_dist, dist, sz, hipMemcpyHostToDevice);
    r_dist = (int*) malloc(sz);

    dim3 grid_phase1(1, 1);
    dim3 grid_phase2(round, 2);
    dim3 grid_phase3(round, round);
    dim3 block(block_size, block_size);

    hipEventRecord(start, 0);
    for (int r = 0; r < round; ++r) {
        kernel_phase1<<<grid_phase1, block>>>(r, n, device_dist);

        kernel_phase2<<<grid_phase2, block>>>(r, n, device_dist);

        kernel_phase3<<<grid_phase3, block>>>(r, n, device_dist);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&k_time, start, stop);
    hipMemcpy(r_dist, device_dist, sz, hipMemcpyDeviceToHost);
    hipFree(device_dist);

    fprintf (stderr, "k_time: %lf\n", k_time);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char* argv[])
{
    block_size = atoi(argv[3]);

    input(argv[1]);
    block_FW(block_size);
    output(argv[2]);
    free(r_dist);
    return 0;
}
