#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include "utils.h"

int n, m, block_size;
int *r_dist;
int world_size, rank;

__global__
void kernel_phase1(int round, int n, int* dist, int bsz)
{
    extern __shared__ int shared_dist[];

    int y = threadIdx.x,
        x = threadIdx.y,
        i = x + round * bsz,
        j = y + round * bsz;

    shared_dist[x * bsz + y] = (i < n && j < n) ? dist[i * n + j] : INF;
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < bsz; ++k) {
        int tmp = shared_dist[x * bsz + k] + shared_dist[k * bsz + y];
        if (tmp < shared_dist[x * bsz + y]) shared_dist[x * bsz + y] = tmp;
        __syncthreads();
    }
    if (i < n && j < n) dist[i * n + j] = shared_dist[x * bsz + y];
    __syncthreads();
}

__global__
void kernel_phase2(int round, int n, int* dist, int bsz)
{
    if (blockIdx.x == round) return;

    extern __shared__ int shared_mem[];
    int* shared_pivot = &shared_mem[0];
    int* shared_dist = &shared_mem[bsz * bsz];

    int y = threadIdx.x,
        x = threadIdx.y,
        i = x + round * bsz,
        j = y + round * bsz;

    shared_pivot[x * bsz + y] = (i < n && j < n) ? dist[i * n + j] : INF;

    if (blockIdx.y == 0)
        j = y + blockIdx.x * bsz;
    else
        i = x + blockIdx.x * bsz;

    if (i >= n || j >= n) return;
    shared_dist[x * bsz + y] = (i < n && j < n) ? dist[i * n + j] : INF;
    __syncthreads();

    if (blockIdx.y == 1) {
        #pragma unroll
        for (int k = 0; k < bsz; ++k) {
            int tmp = shared_dist[x * bsz + k] + shared_pivot[k * bsz + y];
            if (tmp < shared_dist[x * bsz + y]) shared_dist[x * bsz + y] = tmp;
        }
    } else {
        #pragma unroll
        for (int k = 0; k < bsz; ++k) {
            int tmp = shared_pivot[x * bsz + k] + shared_dist[k * bsz + y];
            if (tmp < shared_dist[x * bsz + y]) shared_dist[x * bsz + y] = tmp;
        }
    }

    if (i < n && j < n) dist[i * n + j] = shared_dist[x * bsz + y];
}

__global__
void kernel_phase3(int round, int n, int* dist, int bsz, int offset_lines)
{
    int blockIdx_x = blockIdx.x + offset_lines,
    blockIdx_y = blockIdx.y;
    if (blockIdx_x == round || blockIdx_y == round) return;

    extern __shared__ int shared_mem[];
    int* shared_pivot_row = &shared_mem[0];
    int* shared_pivot_col = &shared_mem[bsz * bsz];

    int y = threadIdx.x,
        x = threadIdx.y,
        i = x + blockIdx_x * blockDim.x,
        j = y + blockIdx_y * blockDim.y,
        i_col = y + round * bsz,
        j_row = x + round * bsz;

    shared_pivot_row[x * bsz + y] = (i < n && i_col < n) ? dist[i * n + i_col] : INF;
    shared_pivot_col[x * bsz + y] = (j < n && j_row < n) ? dist[j_row * n + j] : INF;
    __syncthreads();

    if (i >= n || j >= n) return;
    int dij = dist[i * n + j];
    #pragma unroll
    for (int k = 0; k < bsz; ++k) {
        int tmp = shared_pivot_row[x * bsz + k] + shared_pivot_col[k * bsz + y];
        if (tmp < dij) dij = tmp;
    }
    dist[i * n + j] = dij;
}

__global__
void kernel_swap(int* device_dist, int* swap_dist, int offset_lines, int n)
{
    int blockIdx_x = blockIdx.x + offset_lines;
    int y = threadIdx.x,
        x = threadIdx.y,
        i = x + blockIdx_x * blockDim.x,
        j = y + blockIdx.y * blockDim.y;
    if (i >= n || j >= n) return;
    device_dist[i * n + j] = swap_dist[i * n + j];
}

void block_FW(int block_size)
{
    MPI_Status status;
    float k_time;
    double comm_time = 0, s;

    int round = (n + block_size - 1) / block_size;
    int offset_blks = (round + world_size - 1) / world_size;
    ssize_t sz = sizeof(int) * n * n;

    dim3 grid_phase1(1, 1);
    dim3 grid_phase2(round, 2);
    dim3 grid_phase3((round + world_size - 1) / world_size, round);
    dim3 block(block_size, block_size);

    int *device_dist, *swap_dist, *buffer;
    buffer = (int*) malloc(sz);
    hipSetDevice(rank);
    hipMalloc(&device_dist, sz);
    if (rank == 0) {
        hipMalloc(&swap_dist, sz);
        hipMemcpy(device_dist, dist, sz, hipMemcpyHostToHost);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int r = 0; r < round; ++r) {

        if (rank == 0) {
            kernel_phase1<<<grid_phase1, block, block_size * block_size * sizeof(int)>>>(r, n, device_dist, block_size);
            kernel_phase2<<<grid_phase2, block, block_size * block_size * sizeof(int) * 2>>>(r, n, device_dist, block_size);
            hipStreamSynchronize(0);
            if (round > offset_blks) {
                hipMemcpy(buffer, device_dist, sz, hipMemcpyDeviceToHost);
                s = MPI_Wtime();
                MPI_Send(buffer, sz, MPI_CHAR, 1, 0, MPI_COMM_WORLD);
                comm_time += MPI_Wtime() - s;
            }
        } else if (rank == 1 && round > offset_blks) {
            s = MPI_Wtime();
            MPI_Recv(buffer, sz, MPI_CHAR, 0, 0, MPI_COMM_WORLD, &status);
            comm_time += MPI_Wtime() - s;
            hipMemcpy(device_dist, buffer, sz, hipMemcpyHostToDevice);
        }

        if (rank == 0 || (rank == 1 && round > offset_blks))
            kernel_phase3<<<grid_phase3, block, block_size * block_size * sizeof(int) * 2>>>(r, n, device_dist, block_size, offset_blks * rank);
        hipStreamSynchronize(0);

        if (rank == 0 && round > offset_blks) {
            s = MPI_Wtime();
            MPI_Recv(buffer, sz, MPI_CHAR, 1, 0, MPI_COMM_WORLD, &status);
            comm_time += MPI_Wtime() - s;
            hipMemcpy(swap_dist, buffer, sz, hipMemcpyHostToDevice);
            kernel_swap<<<grid_phase3, block>>>(device_dist, swap_dist, offset_blks, n);
            hipStreamSynchronize(0);
        } else if (rank == 1 && round > offset_blks) {
            hipMemcpy(buffer, device_dist, sz, hipMemcpyDeviceToHost);
            s = MPI_Wtime();
            MPI_Send(buffer, sz, MPI_CHAR, 0, 0, MPI_COMM_WORLD);
            comm_time += MPI_Wtime() - s;
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&k_time, start, stop);
    fprintf (stderr, "k_time: %lf\n", k_time);
    fprintf (stderr, "comm_time: %lf\n", comm_time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    if (rank == 0)
        hipMemcpy(dist, device_dist, sz, hipMemcpyDeviceToHost);
    hipFree(device_dist);
    hipFree(swap_dist);
    r_dist = dist;
}

int main(int argc, char* argv[])
{
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == 0) input(argv[1]);
    MPI_Bcast(&n, 1, MPI_INT, 0, MPI_COMM_WORLD);
    block_size = atoi(argv[3]);

    block_FW(block_size);

    if (rank == 0) output(argv[2]);
    hipHostFree(r_dist);
    MPI_Finalize();
    return 0;
}
